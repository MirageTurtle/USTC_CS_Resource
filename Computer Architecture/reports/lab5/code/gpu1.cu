
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <cstdlib>
#include <ctime>

// #define VERIFY

using namespace std;

__global__ void gemm_baseline(float *A, float *B, float *C, int N);
int idxs2idx(int i, int j, int N);
bool gemm_verify(float *A, float *B, float *C, int N);

int main(int argc, char ** argv)
{
    int n = atoi(argv[1]);
    int N = (1 << n);
    // init
    float *A = new float[N * N];
    float *B = new float[N * N];
    float *C = new float[N * N];
    srand48 (static_cast <unsigned> (time(0)));
    for(int i = 0; i < N * N; i++)
    {
        A[i] = static_cast <float> (drand48());
        B[i] = static_cast <float> (drand48());
    }
    float *dA, *dB, *dC;
    int nBytes = N * N * sizeof(float);
    hipMalloc((void **)&dA, nBytes);
    hipMalloc((void **)&dB, nBytes);
    hipMalloc((void **)&dC, nBytes);
    hipMemcpy((void *)dA, (void *)A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy((void *)dB, (void *)B, nBytes, hipMemcpyHostToDevice);
    int size = 8;
    dim3 block_size(size, size);
    dim3 grid_size((N + size - 1) / size, (N + size - 1) / size);
    gemm_baseline<<<grid_size, block_size>>>(dA, dB, dC, N);
    hipMemcpy((void *)C, (void *)dC, nBytes, hipMemcpyDeviceToHost);
#ifdef VERIFY
    cout << boolalpha << gemm_verify(A, B, C, N) << endl;
#endif
    return 0;
}

int idxs2idx(int i, int j, int N)
{
    return i * N + j;
}

bool gemm_verify(float *A, float *B, float *C, int N)
{
    for(int i = 0; i < N; i++)
    {
        for(int j = 0; j < N; j++)
        {
            float tmp = 0.0f;
            for(int k = 0; k < N; k++)
            {
                tmp += A[idxs2idx(i, k, N)] * B[idxs2idx(k, j, N)];
            }
            // float equal: difference less than 1e-4
            if(abs(tmp - C[idxs2idx(i, j, N)]) >= 1e-4)
            {
                cout << "[WRONG]" << "i: " << i << " j: " << j << " true: " << tmp << " result: " << C[idxs2idx(i, j, N)] << endl;
                cout << "[DIFF]: " << abs(tmp - C[idxs2idx(i, j, N)]) << endl;
                return false;
            }
        }
    }
    return true;
}

__global__ void gemm_baseline(float *A, float *B, float *C, int N)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if(x >= N || y >= N)
    {
        return;
    }
    C[x * N + y] = 0.0f;
    float *pa = A + x * N;
    float *pb = B + y;
    for(int i = 0; i < N; i++, pa++, pb += N)
    {
        C[x * N + y] += (*pa) * (*pb);
    }
    return;
}