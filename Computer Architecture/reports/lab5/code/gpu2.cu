#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cstdlib>
#include <ctime>

// #define VERIFY

using namespace std;

__global__ void blocked_gemm_baseline(float *A, float *B, float *C, int N);
int idxs2idx(int i, int j, int N);
bool gemm_verify(float *A, float *B, float *C, int N);
void print_matrix(float *M, int N);

const int size = (1 << 4);

int main(int argc, char ** argv)
{
    int n = atoi(argv[1]);
    int N = (1 << n);
    // init
    float *A = new float[N * N];
    float *B = new float[N * N];
    float *C = new float[N * N];
    srand48 (static_cast <unsigned> (time(0)));
    for(int i = 0; i < N * N; i++)
    {
        // A[i] = i / 10.0f;
        // B[i] = i / 10.0f;
        A[i] = static_cast <float> (drand48());
        B[i] = static_cast <float> (drand48());
    }

    float *dA, *dB, *dC;
    int nBytes = N * N * sizeof(float);
    hipMalloc((void **)&dA, nBytes);
    hipMalloc((void **)&dB, nBytes);
    hipMalloc((void **)&dC, nBytes);
    hipMemcpy((void *)dA, (void *)A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy((void *)dB, (void *)B, nBytes, hipMemcpyHostToDevice);
    // int size = 8;
    dim3 block_size(size, size);
    dim3 grid_size((N + size - 1) / size, (N + size - 1) / size);
    blocked_gemm_baseline<<<grid_size, block_size>>>(dA, dB, dC, N);
    hipMemcpy((void *)C, (void *)dC, nBytes, hipMemcpyDeviceToHost);
#ifdef VERIFY
    cout << boolalpha << gemm_verify(A, B, C, N) << endl;
#endif
    // print_matrix(A, N);
    // print_matrix(B, N);
    // print_matrix(C, N);
    return 0;
}

int idxs2idx(int i, int j, int N)
{
    return i * N + j;
}

bool gemm_verify(float *A, float *B, float *C, int N)
{
    for(int i = 0; i < N; i++)
    {
        for(int j = 0; j < N; j++)
        {
            float tmp = 0.0f;
            for(int k = 0; k < N; k++)
            {
                tmp += A[idxs2idx(i, k, N)] * B[idxs2idx(k, j, N)];
            }
            // float equal: difference less than 1e-4
            if(abs(tmp - C[idxs2idx(i, j, N)]) >= 1e-4)
            {
                cout << "[WRONG]" << "i: " << i << " j: " << j << " true: " << tmp << " result: " << C[idxs2idx(i, j, N)] << endl;
                cout << "[DIFF]: " << abs(tmp - C[idxs2idx(i, j, N)]) << endl;
                return false;
            }
        }
    }
    return true;
}

__global__ void blocked_gemm_baseline(float *A, float *B, float *C, int N)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if(x >= N || y >= N)
    {
        return;
    }
    
    int tmp_x = threadIdx.x;
    int tmp_y = threadIdx.y;
    int block_num = (N + blockDim.x - 1) / blockDim.x;

    // const int block_size = (1 << 3);
    const int block_size = size;

    __shared__ float blockA[block_size][block_size];
    __shared__ float blockB[block_size][block_size];
    int A_start = blockIdx.x * block_size * N;
    int B_start = blockIdx.y * block_size;
    int A_step = block_size;
    int B_step = block_size * N;

    // 使用tmp减少与数组的交互，提升速度
    // 矩阵规模为 2^13 时，可以从2s+提升到1s+
    float tmp = 0.0f;
    for(int i = 0; i < block_num; i++)
    {
        blockA[tmp_x][tmp_y] = A[A_start + i * A_step + tmp_x * N + tmp_y];
        blockB[tmp_x][tmp_y] = B[B_start + i * B_step + tmp_x * N + tmp_y];
        __syncthreads();
        for(int j = 0; j < blockDim.x; j++)
        {
            // C[x * N + y] += blockA[tmp_x][j] * blockB[j][tmp_y];
            tmp += blockA[tmp_x][j] * blockB[j][tmp_y];
        }
        __syncthreads();
    }
    C[x * N + y] = tmp;
}

void print_matrix(float *M, int N)
{
    for(int i = 0; i < N; i++)
    {
        for(int j = 0; j < N; j++)
        {
            cout << M[idxs2idx(i, j, N)] << " ";
        }
        cout << endl;
    }
    return;
}